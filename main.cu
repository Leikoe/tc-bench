// very heavily modified https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuBLASLt/LtFp8Matmul/sample_cublasLt_LtFp8Matmul.cu
#include <time.h>
#include <stdio.h>
#include <hipblaslt.h>
#include "helpers.h"

#define ITERS 1000
#define N 4096
#define TILE_SIZE 16
#define WARP_SIZE 32

uint64_t nanos()
{
    struct timespec start;
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    return (uint64_t)start.tv_sec * 1000000000 + (uint64_t)start.tv_nsec;
}

int main()
{
    srand(time(NULL));

    hipblasLtHandle_t ltHandle;
    hipblasLtCreate(&ltHandle);

    int m = N;
    int k = N;
    int n = N;

    unsigned char *A, *B;
    int *C;
    hipMalloc(&A, N * N * sizeof(unsigned char));
    hipMalloc(&B, N * N * sizeof(unsigned char));
    hipMalloc(&C, N * N * sizeof(int));

    size_t workspaceSize = 1024 * 1024 * 4;
    void *workspace;
    hipMalloc(&workspace, workspaceSize);

    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasOperation_t transb = HIPBLAS_OP_T;
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32I, HIP_R_32I));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8I, m, k, k));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8I, k, n, n));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32I, m, n, n));

    hipblasLtMatmulPreference_t preference = NULL;
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0)
    {
        checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    // Scaling factors
    int32_t alpha = 1;
    int32_t beta = 0;

    uint64_t start = nanos();

    for (int i = 0; i < ITERS; i++)
    {
        checkCublasStatus(hipblasLtMatmul(ltHandle,
                                         operationDesc,
                                         &alpha,
                                         A,
                                         Adesc,
                                         B,
                                         Bdesc,
                                         &beta,
                                         C,
                                         Cdesc,
                                         C,
                                         Cdesc,
                                         &heuristicResult.algo,
                                         workspace,
                                         workspaceSize,
                                         0));
    }

    hipDeviceSynchronize();
    uint64_t end = nanos();

    double gflop = (2.0 * N * N * N) * 1e-9 * (float)ITERS;
    double s = (end - start) * 1e-9;
    printf("%f TOPS -- %.2f ms\n", (gflop / 1000.) / s, s * 1e3);

    checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipFree(workspace);
}
